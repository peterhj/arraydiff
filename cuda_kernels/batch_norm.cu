
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdint.h>

#define OFFSET_BANK(idx) ({ __typeof__ (idx) _idx = idx; ((_idx) + ((_idx) / 32)); })

__global__ void conv_diag_affine_white_var_fwd_batch_kernel(
    uint32_t spatial_dim,
    uint32_t chan_dim,
    uint32_t batch_sz,
    const float *x,
    const float *mean,
    const float *var,
    float epsilon,
    float *y)
{
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  uint32_t u = idx % spatial_dim;
  uint32_t c = (idx / spatial_dim) % chan_dim;
  uint32_t batch_idx = idx / (spatial_dim * chan_dim);
  if (u < spatial_dim && c < chan_dim && batch_idx < batch_sz) {
    float m = mean[c];
    float v = var[c];
    float y_i = (x[idx] - m) * rsqrtf(v + epsilon);
    y[idx] = y_i;
  }
}

extern "C" void arraydiff_cuda_kernel_conv_normalize_fwd_f32(
    size_t spatial_dim,
    size_t chan_dim,
    size_t batch_sz,
    const float *x,
    const float *mean,
    const float *var,
    float epsilon,
    float *y,
    hipStream_t stream)
{
  uint32_t n = spatial_dim * chan_dim * batch_sz;
  conv_diag_affine_white_var_fwd_batch_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(
      spatial_dim, chan_dim, batch_sz, x, mean, var, epsilon, y);
}

__global__ void conv_normalize_var_bwd_nonatomic_f32_kernel(
    uint32_t round_offset,
    uint32_t spatial_dim,
    uint32_t chan_dim,
    uint32_t batch_sz,
    const float *x,
    const float *mean,
    const float *var,
    const float *y_grad,
    float epsilon,
    float *var_grad)
{
  __shared__ float cache[1024+32];
  uint32_t block_dim = min(blockDim.x, spatial_dim * batch_sz - round_offset);
  uint32_t round_idx = round_offset + threadIdx.x;
  uint32_t spatial_idx = round_idx % spatial_dim;
  uint32_t batch_idx = round_idx / spatial_dim;
  uint32_t chan_idx = blockIdx.x;
  if (spatial_idx < spatial_dim && chan_idx < chan_dim && batch_idx < batch_sz) {
    uint32_t idx = spatial_idx + spatial_dim * (chan_idx + chan_dim * batch_idx);
    float v = var[chan_idx];
    cache[OFFSET_BANK(threadIdx.x)] = -0.5f * y_grad[idx] * (x[idx] - mean[chan_idx]) * rsqrtf(v + epsilon) / (v + epsilon);
  } else {
    cache[OFFSET_BANK(threadIdx.x)] = 0.0f;
  }
  __syncthreads();
  for (uint32_t s = 1; s < blockDim.x; s *= 2) {
    if (spatial_idx < spatial_dim && chan_idx < chan_dim && batch_idx < batch_sz) {
      if ((threadIdx.x & (2 * s - 1)) == 0 && (threadIdx.x + s) < block_dim) {
        cache[OFFSET_BANK(threadIdx.x)] += cache[OFFSET_BANK(threadIdx.x + s)];
      }
    }
    __syncthreads();
  }
  if (spatial_idx < spatial_dim && chan_idx < chan_dim && batch_idx < batch_sz) {
    if (threadIdx.x == 0) {
      var_grad[chan_idx] += cache[0];
    }
  }
}

__global__ void conv_normalize_mean_bwd_nonatomic_f32_kernel(
    uint32_t round_offset,
    uint32_t spatial_dim,
    uint32_t chan_dim,
    uint32_t batch_sz,
    const float *x,
    const float *mean,
    const float *var,
    const float *var_grad,
    const float *y_grad,
    float epsilon,
    float *mean_grad)
{
  __shared__ float cache[1024+32];
  uint32_t block_dim = min(blockDim.x, spatial_dim * batch_sz - round_offset);
  uint32_t round_idx = round_offset + threadIdx.x;
  uint32_t spatial_idx = round_idx % spatial_dim;
  uint32_t batch_idx = round_idx / spatial_dim;
  uint32_t chan_idx = blockIdx.x;
  if (spatial_idx < spatial_dim && chan_idx < chan_dim && batch_idx < batch_sz) {
    uint32_t idx = spatial_idx + spatial_dim * (chan_idx + chan_dim * batch_idx);
    cache[OFFSET_BANK(threadIdx.x)] = -(y_grad[idx] * rsqrtf(var[chan_idx] + epsilon) + 2.0f * var_grad[chan_idx] * (x[idx] - mean[chan_idx]) / ((float)(spatial_dim * (batch_sz - 1))));
  } else {
    cache[OFFSET_BANK(threadIdx.x)] = 0.0f;
  }
  __syncthreads();
  for (uint32_t s = 1; s < blockDim.x; s *= 2) {
    if (spatial_idx < spatial_dim && chan_idx < chan_dim && batch_idx < batch_sz) {
      if ((threadIdx.x & (2 * s - 1)) == 0 && (threadIdx.x + s) < block_dim) {
        cache[OFFSET_BANK(threadIdx.x)] += cache[OFFSET_BANK(threadIdx.x + s)];
      }
    }
    __syncthreads();
  }
  if (spatial_idx < spatial_dim && chan_idx < chan_dim && batch_idx < batch_sz) {
    if (threadIdx.x == 0) {
      mean_grad[chan_idx] += cache[0];
    }
  }
}

__global__ void conv_normalize_x_bwd_nonatomic_f32_kernel(
    uint32_t spatial_dim,
    uint32_t chan_dim,
    uint32_t batch_sz,
    const float *var,
    const float *y_grad,
    float epsilon,
    float *x_grad)
{
  uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
  uint32_t spatial_idx = idx % spatial_dim;
  uint32_t chan_idx = (idx / spatial_dim) % chan_dim;
  uint32_t batch_idx = (idx / spatial_dim) / chan_dim;
  if (spatial_idx < spatial_dim && chan_idx < chan_dim && batch_idx < batch_sz) {
    x_grad[idx] += y_grad[idx] * rsqrtf(var[chan_idx] + epsilon);
  }
}

__global__ void conv_batch_mean_fwd_nonatomic_f32_kernel(
    uint32_t round_offset,
    uint32_t spatial_dim,
    uint32_t chan_dim,
    uint32_t batch_sz,
    const float *x,
    float *mean)
{
  __shared__ float cache[1024+32];
  uint32_t block_dim = min(blockDim.x, spatial_dim * batch_sz - round_offset);
  uint32_t round_idx = round_offset + threadIdx.x;
  uint32_t spatial_idx = round_idx % spatial_dim;
  uint32_t batch_idx = round_idx / spatial_dim;
  uint32_t chan_idx = blockIdx.x;
  if (spatial_idx < spatial_dim && chan_idx < chan_dim && batch_idx < batch_sz) {
    uint32_t idx = spatial_idx + spatial_dim * (chan_idx + chan_dim * batch_idx);
    cache[OFFSET_BANK(threadIdx.x)] = x[idx];
  } else {
    cache[OFFSET_BANK(threadIdx.x)] = 0.0f;
  }
  __syncthreads();
  for (uint32_t s = 1; s < blockDim.x; s *= 2) {
    if (spatial_idx < spatial_dim && chan_idx < chan_dim && batch_idx < batch_sz) {
      if ((threadIdx.x & (2 * s - 1)) == 0 && (threadIdx.x + s) < block_dim) {
        cache[OFFSET_BANK(threadIdx.x)] += cache[OFFSET_BANK(threadIdx.x + s)];
      }
    }
    __syncthreads();
  }
  if (spatial_idx < spatial_dim && chan_idx < chan_dim && batch_idx < batch_sz) {
    if (threadIdx.x == 0) {
      mean[chan_idx] += cache[0] / ((float)(spatial_dim * batch_sz));
    }
  }
}

extern "C" void arraydiff_cuda_kernel_conv_batch_mean_fwd_nonatomic_f32(
    size_t spatial_dim,
    size_t chan_dim,
    size_t batch_sz,
    const float *x,
    float *mean,
    hipStream_t stream)
{
  // XXX: `mean` should be zeroed.
  uint32_t num_rounds = (spatial_dim * batch_sz + 1024-1) / 1024;
  uint32_t num_blocks = chan_dim;
  for (uint32_t round = 0; round < num_rounds; round++) {
    conv_batch_mean_fwd_nonatomic_f32_kernel<<<num_blocks, 1024, 0, stream>>>(
        round * 1024, spatial_dim, chan_dim, batch_sz, x, mean);
  }
}

__global__ void conv_batch_var_fwd_nonatomic_f32_kernel(
    uint32_t round_offset,
    uint32_t spatial_dim,
    uint32_t chan_dim,
    uint32_t batch_sz,
    const float *x,
    const float *mean,
    float *var)
{
  __shared__ float cache[1024+32];
  uint32_t block_dim = min(blockDim.x, spatial_dim * batch_sz - round_offset);
  uint32_t round_idx = round_offset + threadIdx.x;
  uint32_t spatial_idx = round_idx % spatial_dim;
  uint32_t batch_idx = round_idx / spatial_dim;
  uint32_t chan_idx = blockIdx.x;
  if (spatial_idx < spatial_dim && chan_idx < chan_dim && batch_idx < batch_sz) {
    uint32_t idx = spatial_idx + spatial_dim * (chan_idx + chan_dim * batch_idx);
    float residual = x[idx] - mean[chan_idx];
    cache[OFFSET_BANK(threadIdx.x)] = residual * residual;
  } else {
    cache[OFFSET_BANK(threadIdx.x)] = 0.0f;
  }
  __syncthreads();
  for (uint32_t s = 1; s < blockDim.x; s *= 2) {
    if (spatial_idx < spatial_dim && chan_idx < chan_dim && batch_idx < batch_sz) {
      if ((threadIdx.x & (2 * s - 1)) == 0 && (threadIdx.x + s) < block_dim) {
        cache[OFFSET_BANK(threadIdx.x)] += cache[OFFSET_BANK(threadIdx.x + s)];
      }
    }
    __syncthreads();
  }
  if (spatial_idx < spatial_dim && chan_idx < chan_dim && batch_idx < batch_sz) {
    if (threadIdx.x == 0) {
      var[chan_idx] += cache[0] / ((float)(spatial_dim * (batch_sz - 1)));
    }
  }
}

extern "C" void arraydiff_cuda_kernel_conv_batch_var_fwd_nonatomic_f32(
    size_t spatial_dim,
    size_t chan_dim,
    size_t batch_sz,
    const float *x,
    const float *mean,
    float *var,
    hipStream_t stream)
{
  // XXX: `var` should be zeroed.
  uint32_t num_rounds = (spatial_dim * batch_sz + 1024-1) / 1024;
  uint32_t num_blocks = chan_dim;
  for (uint32_t round = 0; round < num_rounds; round++) {
    conv_batch_var_fwd_nonatomic_f32_kernel<<<num_blocks, 1024, 0, stream>>>(
        round * 1024, spatial_dim, chan_dim, batch_sz, x, mean, var);
  }
}

__global__ void conv_batch_stats_bwd_f32_kernel(
    uint32_t spatial_dim,
    uint32_t chan_dim,
    uint32_t batch_sz,
    const float *x,
    const float *mean,
    const float *mean_grad,
    const float *var_grad,
    float epsilon,
    float *x_grad)
{
  uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
  uint32_t spatial_idx = idx % spatial_dim;
  uint32_t chan_idx = (idx / spatial_dim) % chan_dim;
  uint32_t batch_idx = (idx / spatial_dim) / chan_dim;
  if (spatial_idx < spatial_dim && chan_idx < chan_dim && batch_idx < batch_sz) {
    x_grad[idx] += mean_grad[chan_idx] / ((float)(spatial_dim * batch_sz)) + 2.0f * var_grad[chan_idx] * (x[idx] - mean[chan_idx]) / ((float)(spatial_dim * (batch_sz - 1)));;
  }
}